#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ 
void vecAddKernel(float *A, float *B, float *C, int n)
{
   int i = threadIdx.x + blockDim.x * blockIdx.x;
   if (i < n)
      C[i] = A[i] + B[i];
}

__global__
void vecAddKernel(float *A, float *B, float *C, int dimension)
{
   int id = threadIdx.x + blockDim.x * blockIdx.x;
   int i;
   for (i = id; i < id + dimension; i++)
      if (i < dimension * dimension)
         C[i] = A[i] + B[i];

}


__host__ 
void vecAdd(float *h_A, float *h_B, float* h_C, int n)
{
   int size = n * sizeof(float);
   float *d_A, *d_B, *d_C;

   hipMalloc((void **) &d_A, size);
   hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice); 
   hipMalloc((void **) &d_B, size);
   hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice); 

   hipMalloc((void **) &d_C, size);

   vecAddKernel<<<ceil(n / 256.0), 256>>>(d_A, d_B, d_C, n);

   hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
}

int main(int argc, char *argv[])
{
   int n = atoi(argv[1]);
   printf("Size: %d\n", n);

   float *h_A = (float *) malloc(n * sizeof(float));
   float *h_B = (float *) malloc(n * sizeof(float));
   float *h_C = (float *) malloc(n * sizeof(float));

   int i;
   for (i = 0; i < n; i++)
   {
      h_A[i] = 1;
      h_B[i] = 1;
   }

   vecAdd(h_A, h_B, h_C, n);  
 
   printf("%.2f\n", h_C[0]);

   free(h_A);
   free(h_B);
   free(h_C);
   
   return 0;
}
