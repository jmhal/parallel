#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__
void vecAddKernel(float *A, float *B, float *C, int n)
{
   int id = threadIdx.x + blockDim.x * blockIdx.x;
   int i;
   for (i = id; i <= n * (n - 1) + id; i += n)
      if (i < n * n)
         C[i] = A[i] + B[i];
}


__host__ 
void vecAdd(float *h_A, float *h_B, float* h_C, int n)
{
   int size = n * n * sizeof(float);
   float *d_A, *d_B, *d_C;

   hipMalloc((void **) &d_A, size);
   hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice); 
   hipMalloc((void **) &d_B, size);
   hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice); 

   hipMalloc((void **) &d_C, size);

   vecAddKernel<<<ceil(n / 256.0), 256>>>(d_A, d_B, d_C, n);

   hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
}

int main(int argc, char *argv[])
{
   int n = atoi(argv[1]);
   printf("Size: %d\n", n);

   float *h_A = (float *) malloc(n * n * sizeof(float));
   float *h_B = (float *) malloc(n * n * sizeof(float));
   float *h_C = (float *) malloc(n * n * sizeof(float));

   int i;
   for (i = 0; i < n * n; i++)
   {
      h_A[i] = 1;
      h_B[i] = 1;
   }

   vecAdd(h_A, h_B, h_C, n);  
 
   printf("%.2f\n", h_C[100]);

   free(h_A);
   free(h_B);
   free(h_C);
   
   return 0;
}
