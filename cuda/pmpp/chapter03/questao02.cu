#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__
void vecMatMultKernel(float *A, float *B, float *C, int n)
{
   int i = threadIdx.x + blockDim.x * blockIdx.x;
   A[i] = 0;
   int j;
   for (j = 0; j < n; j++)  
   {
      if (i * n + j < n * n)
         A[i] += B[i * n + j] * C[j];
         // A[i] += B[i * n + j];
   } 
}


__host__ 
void vecMatMult(float *h_A, float *h_B, float* h_C, int n)
{
   int size = n * sizeof(float);
   float *d_A, *d_B, *d_C;

   
   hipMalloc((void **) &d_B, n * size);
   hipMemcpy(d_B, h_B, n * size, hipMemcpyHostToDevice); 
   hipMalloc((void **) &d_C, size);
   hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice); 

   hipMalloc((void **) &d_A, size);

   vecMatMultKernel<<<ceil(n / 256.0), 256>>>(d_A, d_B, d_C, n);

   hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
}

int main(int argc, char *argv[])
{
   int n = atoi(argv[1]);
   printf("Dimension: %d\n", n);

   float *h_B = (float *) malloc(n * n * sizeof(float));
   float *h_C = (float *) malloc(n * sizeof(float));
   float *h_A = (float *) malloc(n * sizeof(float));

   int i;
   for (i = 0; i < n * n; i++)
   {
      h_B[i] = 1;
      if (i < n)
         h_C[i] = 1;
   }

   vecMatMult(h_A, h_B, h_C, n);  
 
  
   for (i = 0; i < n; i++) 
   {
      printf("%.2f ", h_A[i]);
      if (!((i+1) % 10))
         printf("\n");
   }

   printf("\n");

   free(h_A);
   free(h_B);
   free(h_C);
   
   return 0;
}
